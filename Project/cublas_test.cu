#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CHECK_CUDA(call) \
    if ((call) != hipSuccess) { \
        std::cerr << "CUDA error at " << __LINE__ << std::endl; exit(EXIT_FAILURE); \
    }

#define CHECK_CUBLAS(call) \
    if ((call) != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "cuBLAS error at " << __LINE__ << std::endl; exit(EXIT_FAILURE); \
    }

int main() {
    const int M = 2, K = 3, N = 3;  // A: MxK, B: KxN => C: MxN (row-major)
    
    float h_A[M*K] = {1, 2, 3,   // A = [ [1,2,3],
                       4, 5, 6};  //       [4,5,6] ]
    
    float h_B[K*N] = {1, 1, 1,     // B = [ [1,4],
                      1, 1, 1,     //       [2,5],
                      1, 1, 1};     //       [3,6] ]
    
    float h_C[M*N] = {0};         // Output matrix

    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, sizeof(float) * M * K));
    CHECK_CUDA(hipMalloc(&d_B, sizeof(float) * K * N));
    CHECK_CUDA(hipMalloc(&d_C, sizeof(float) * M * N));

    CHECK_CUDA(hipMemcpy(d_A, h_A, sizeof(float) * M * K, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, sizeof(float) * K * N, hipMemcpyHostToDevice));

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    float alpha = 1.0f;
    float beta = 0.0f;

    // A and B are row-major, simulate by transposing both
    CHECK_CUBLAS(hipblasSgemm(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        N, M, K,           // Result is MxN, so rows=N, cols=M in column-major
        &alpha,
        d_B, N,            // B is KxN in row-major, becomes NxK column-major
        d_A, K,            // A is MxK in row-major, becomes KxM column-major
        &beta,
        d_C, N             // C is MxN in row-major, store as N x M column-major
    ));

    CHECK_CUDA(hipMemcpy(h_C, d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost));

    std::cout << "Result C = A * B:\n";
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            std::cout << h_C[i * N + j] << " ";
        }
        std::cout << "\n";
    }

    // Cleanup
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}
